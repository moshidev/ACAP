#include "hip/hip_runtime.h"
﻿#include <assert.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>     // srand, rand
#include <time.h>       // time
#include <locale.h>
#ifdef WIN32
#include <Windows.h>
int gettimeofday(struct timeval* tp, struct timezone* tzp)
{
    // Note: some broken versions only have 8 trailing zero's, the correct epoch has 9 trailing zero's
    // This magic number is the number of 100 nanosecond intervals since January 1, 1601 (UTC)
    // until 00:00:00 January 1, 1970 
    static const uint64_t EPOCH = ((uint64_t)116444736000000000ULL);

    SYSTEMTIME  system_time;
    FILETIME    file_time;
    uint64_t    time;

    GetSystemTime(&system_time);
    SystemTimeToFileTime(&system_time, &file_time);
    time = ((uint64_t)file_time.dwLowDateTime);
    time += ((uint64_t)file_time.dwHighDateTime) << 32;

    tp->tv_sec = (long)((time - EPOCH) / 10000000L);
    tp->tv_usec = (long)(system_time.wMilliseconds * 1000);
    return 0;
}
#else
#include <sys/time.h>
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define IMG_DEPTH 256

struct parametros {
    size_t niter;
    size_t img_len;
    uint16_t nblocks;
    uint16_t nthreads_per_block;
};

typedef struct {
    uint8_t* v;
    size_t len_i;
    size_t len_j;
} img_t;

static double get_wall_time() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

static img_t rand_1D_img(size_t len_j) {
    srand(0);
    img_t img = {(uint8_t*)malloc(len_j), 1, len_j};
    for (unsigned long i = 0; i < len_j; i++) {
        img.v[i] = rand() % IMG_DEPTH;
    }
    return img;
}

static void histogramaCPU(unsigned char* img, size_t nBytes, unsigned int* histo) {
    double t1 = get_wall_time();
    for (int i = 0; i < IMG_DEPTH; i++)
        histo[i] = 0;//Inicializacion
    for (size_t i = 0; i < nBytes; i++) {
        histo[img[i]]++;
    }
    double t2 = get_wall_time();
    printf("Tiempo de CPU (s): %.4lf\n", t2-t1);
}

__global__ void kernelHistograma(unsigned char* imagen, size_t size, unsigned int* ghisto) {
    __shared__ unsigned int lhisto[IMG_DEPTH];
    for (unsigned t = threadIdx.x; t < IMG_DEPTH; t += blockDim.x) {
        lhisto[t] = 0;
        t += blockDim.x;
    }

    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    __syncthreads();
    while (i < size) {
        atomicAdd(&lhisto[imagen[i]], 1);
        i += offset;
    }

    __syncthreads();
    for (unsigned t = threadIdx.x; t < IMG_DEPTH; t += blockDim.x) {
        atomicAdd(&ghisto[t], lhisto[t]);
    }
}

static void assert_good_algorithm_histogram_cuda(struct parametros p) {
    unsigned char* imagen = (unsigned char*)rand_1D_img(p.img_len).v;

    /* Calcula el histograma en la CPU */
    unsigned int histoCPU[IMG_DEPTH];
    histogramaCPU(imagen, p.img_len, histoCPU);

    /* Calcula el histograma en la GPU */
    unsigned int histoGPU[IMG_DEPTH];
    unsigned char* dev_imagen = 0;
    unsigned int* dev_histo = 0;
    hipMalloc((void**)&dev_imagen, p.img_len);
    hipMemcpy(dev_imagen, imagen, p.img_len, hipMemcpyHostToDevice);
    hipMalloc((void**)&dev_histo, IMG_DEPTH * sizeof(unsigned int));
    hipMemset(dev_histo, 0, IMG_DEPTH * sizeof(unsigned int));
    kernelHistograma << <p.nblocks, p.nthreads_per_block >> > (dev_imagen, p.img_len, dev_histo);
    hipMemcpy(histoGPU, dev_histo, IMG_DEPTH * sizeof(unsigned int), hipMemcpyDeviceToHost);

    /* Comprueba que son el mismo histograma */
    assert(memcmp(histoCPU, histoGPU, IMG_DEPTH) == 0);
    printf("Calculo correcto!!\n");
    hipFree(dev_imagen);
    hipFree(dev_histo);
    free(imagen);
}

struct parametros get_params(int argc, char** argv) {
    struct parametros p;
    const char txt_parametros[] = "<número bloques> <hebras por bloque> <número de iteraciones del benchmark> <tamaño de la imagen en bytes>";
    if (argc != 5) {
        fprintf(stderr, "No puedo ejecutar el programa si no me indicas qué tengo que hacer.\n");
        fprintf(stderr, "Uso: %s %s.\n", argv[0], txt_parametros);
        exit(1);
    }
    else {
        unsigned long nblocks = strtoul(argv[1], NULL, 0);
        unsigned long nthreads_per_block = strtoul(argv[2], NULL, 0);
        unsigned long niter = strtoul(argv[3], NULL, 0);
        unsigned long img_len = strtoul(argv[4], NULL, 0);
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        if (nblocks > UINT16_MAX || nthreads_per_block > (unsigned)prop.maxThreadsPerBlock) {
            fprintf(stderr, "Parámetros de ejecución incompatibles con la tarjeta gráfica.\n");
            fprintf(stderr, "Número de bloques esperados: menor igual que %d ; obtenidos: %d\n", UINT16_MAX, nblocks);
            fprintf(stderr, "Número de hilos por bloque esperados: menor igual que %d ; obtenidos: %d\n", prop.maxThreadsPerBlock, nthreads_per_block);
            fprintf(stderr, "Uso: %s %s.\n", argv[0], txt_parametros);
            exit(1);
        }
        p.nblocks = (uint16_t)nblocks;
        p.nthreads_per_block = (uint16_t)nthreads_per_block;
        p.niter = (size_t)niter;
        p.img_len = (size_t)img_len;
    }
    return p;
}

int main(int argc, char** argv) {
    /* Comprueba los parámetros introducidos, abortando si no son correctos */
    setlocale(LC_ALL, "");
    struct parametros p = get_params(argc, argv);

    /* Comprueba que la implementación CUDA es válida */
    assert_good_algorithm_histogram_cuda(p);

    /* Inicializa la memoria del dispositivo */
    unsigned char* imagen = (unsigned char*)rand_1D_img(p.img_len).v;
    hipEvent_t global_start, global_end;
    hipEventCreate(&global_start);
    hipEventCreate(&global_end);
    hipDeviceSynchronize();
    hipEventRecord(global_start);
    unsigned char* dev_imagen = 0;
    unsigned int* dev_histo = 0;
    hipMalloc((void**)&dev_imagen, p.img_len);
    hipMemcpy(dev_imagen, imagen, p.img_len, hipMemcpyHostToDevice);
    hipMalloc((void**)&dev_histo, IMG_DEPTH * sizeof(unsigned int));

    /* Inicializa los recursos para el benchmark */
    hipEvent_t iter_start, iter_end;
    hipEventCreate(&iter_start);
    hipEventCreate(&iter_end);
    float gpu_compute_ms = 0.0f;

    /* Ejecuta el benchmark tantas veces como se indique */
    if (p.niter > 0) {
        hipMemset(dev_histo, 0, IMG_DEPTH * sizeof(unsigned int));
        kernelHistograma << <p.nblocks, p.nthreads_per_block >> > (dev_imagen, p.img_len, dev_histo);  // la primera iteración suele tardar más que las posteriores
    }
    for (int iter = 0; iter < p.niter-1; iter++) {
        hipMemset(dev_histo, 0, IMG_DEPTH * sizeof(unsigned int));
        hipDeviceSynchronize();
        hipEventRecord(iter_start);
        kernelHistograma << <p.nblocks, p.nthreads_per_block >> > (dev_imagen, p.img_len, dev_histo);
        hipEventRecord(iter_end);
        hipEventSynchronize(iter_end);

        float elapsed_ms;
        hipEventElapsedTime(&elapsed_ms, iter_start, iter_end);
        gpu_compute_ms += elapsed_ms;
    }

    /* Trae a memoria local el último histograma de la última ejecución del benchmark */
    uint32_t histoGPU[IMG_DEPTH];
    hipMemcpy(histoGPU, dev_histo, IMG_DEPTH * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipEventRecord(global_end);
    hipEventSynchronize(global_end);

    /* Imprime por pantalla el tiempo medio de ejecución por benchmark */
    float gpu_avg_compute_seconds_per_iter = gpu_compute_ms / (p.niter * 1000.0f);
    printf("Tiempo medio de ejecucion del kernel<<<%d, %d>>> sobre %zu bytes [s]: %.4f\n", p.nblocks, p.nthreads_per_block, p.img_len, gpu_avg_compute_seconds_per_iter);
    float elapsed_ms;
    hipEventElapsedTime(&elapsed_ms, global_start, global_end);
    float gpu_global_compute_seconds = elapsed_ms / 1000.0f;
    printf("Tiempo total de ejecución del benchmark del kernel<<<%d, %d>>> sobre %zu bytes [s]: %.4f\n", p.nblocks, p.nthreads_per_block, p.img_len, gpu_global_compute_seconds);

    /* Destruye los recursos utilizados */
    hipEventDestroy(global_start);
    hipEventDestroy(global_end);
    hipEventDestroy(iter_start);
    hipEventDestroy(iter_end);
    hipFree(dev_imagen);
    hipFree(dev_histo);
    free(imagen);

    return 0;
}