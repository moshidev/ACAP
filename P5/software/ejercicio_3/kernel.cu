#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <stdint.h>
#include <locale.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#ifdef WIN32
#include <Windows.h>
int gettimeofday(struct timeval* tp, struct timezone* tzp)
{
    // Note: some broken versions only have 8 trailing zero's, the correct epoch has 9 trailing zero's
    // This magic number is the number of 100 nanosecond intervals since January 1, 1601 (UTC)
    // until 00:00:00 January 1, 1970 
    static const uint64_t EPOCH = ((uint64_t)116444736000000000ULL);

    SYSTEMTIME  system_time;
    FILETIME    file_time;
    uint64_t    time;

    GetSystemTime(&system_time);
    SystemTimeToFileTime(&system_time, &file_time);
    time = ((uint64_t)file_time.dwLowDateTime);
    time += ((uint64_t)file_time.dwHighDateTime) << 32;

    tp->tv_sec = (long)((time - EPOCH) / 10000000L);
    tp->tv_usec = (long)(system_time.wMilliseconds * 1000);
    return 0;
}
#else
#include <sys/time.h>
#endif

static double get_wall_time() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

#define NULL_PTR    0
#define STRTOL_INFER_BASE   0

typedef struct matrix {
    size_t n;
    size_t m;
    double* v;
} matrix_t;

#define acc(matrix, i, j) \
    ((matrix).v[i*(matrix).m+j])

int matrix_alloc(size_t n, size_t m, matrix_t* matrix) {
    double* vector = (double*)calloc(n*m, sizeof(double));
    if (vector) {
        matrix->n = n;
        matrix->m = m;
        matrix->v = vector;
        return 0;
    }
    return -1;
}

void matrix_destroy(matrix_t* matrix) {
    matrix->n = 0;
    matrix->m = 0;
    free(matrix->v);
    matrix->v = 0;
}

void matrix_print(matrix_t m) {
    for (size_t i = 0; i < m.n; i++) {
        for (size_t j = 0; j < m.m; j++) {
            printf("%.2f", acc(m, i, j));
            putchar(j != m.m-1 ? '\t' : '\n');
        }
    }
}

int matrix_mul(matrix_t A, matrix_t B, matrix_t* dst) {
    if (A.m != B.n) {
        return -1;
    }
    int errDst = matrix_alloc(A.n, B.m, dst);
    if (errDst != 0) {
        return -2;
    }

    const size_t suml = A.m;
    for (size_t i = 0; i < dst->n; i++) {
        for (size_t j = 0; j < dst->m; j++) {
            double sum = 0.0;
            for (size_t w = 0; w < suml; w++) {
                sum += acc(A, i, w) * acc(B, w, j);
            }
            acc(*dst, i, j) = sum;
        }
        /*if (i % (dst->n/100) == 0) {
            printf("%.2f%%\r", 100.0*i/dst->n);
            fflush(stdout);
        }*/
    }

    return 0;
}

void randomize(double* v, size_t v_len) {
    int seed = time(NULL_PTR);
    srand(seed);
    printf("Semilla utilizada: %d\n", seed);
    size_t p = 0;
    for (size_t i = 0; i < v_len; i++) {
        if (p >= 100000000) {
            p = 0;
            printf("Inicializados %zu elementos aleatorios (%f%%).\r", i, 100.0*((double)i)/v_len);
        }
        ++p;
        v[i] = rand()*rand();
    }
    printf("Inicializados %zu elementos aleatorios (%f%%).\n", v_len, 100.0);
}

void memExit39(void) {
    fprintf(stderr, "ª >w< son demasiados datos! no he podido reservar un trozo de memoria contigua tan grande!\n");
    fprintf(stderr, "Error fatal\n");
    exit(39);
}

__global__ void kernelMatrixMul(double* va, double* vb, size_t n, size_t m, size_t l, double* dst) {
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;

    while (i < n) {
        for (size_t j = 0; j < l; j++) {
            double sum = 0.0;
            for (size_t w = 0; w < m; w++) {
                sum += va[i * m + w] * vb[w * l + j];
            }
            dst[i * l + j] = sum;
        }
        i += blockDim.x * gridDim.x;
    }
}

int main(int argc, char** argv) {
    setlocale(LC_ALL, "");

    /* Inicializamos dos matrices */
    matrix_t A, B;
    int errA = matrix_alloc(1000, 500, &A);
    int errB = matrix_alloc(500, 700, &B);
    if (errA != 0 || errB != 0) {
        memExit39();
    }
    //randomize(A.v, A.n*A.m);
    //randomize(B.v, B.n*B.m);
    for (size_t i = 0; i < A.n*A.m; i++) {
        A.v[i] = i%5+1;
    }
    for (size_t i = 0; i < B.n * B.m; i++) {
        B.v[i] = i%5+1;
    }

    /* Multiplicamos ambas matrices en la CPU */
    double cpu_s = get_wall_time();
    matrix_t productAB;
    int errMul = matrix_mul(A, B, &productAB);
    if (errMul != 0) {
        memExit39();
    }
    if (productAB.n*productAB.m < 9*9) {
        printf("Matriz A:\n");
        matrix_print(A);
        printf("Matriz B:\n");
        matrix_print(B);
        printf("Producto de ambas matrices (CPU):\n");
        matrix_print(productAB);
    }
    cpu_s = get_wall_time() - cpu_s;
    printf("t_cpu_s:%.8f\n", cpu_s);

    /* Monitoriza */
    hipEvent_t global_start, global_end;
    hipEventCreate(&global_start);
    hipEventCreate(&global_end);
    hipEventRecord(global_start);

    /* Multiplicamos ambas matrices en la GPU */
    double *dev_va, *dev_vb, *dev_vdst;
    int nthreads = 512, nblocks = 3070;
    hipMalloc(&dev_va, A.n * A.m * sizeof(double));
    hipMalloc(&dev_vb, B.n * B.m * sizeof(double));
    hipMalloc(&dev_vdst, A.n * B.m * sizeof(double));

    hipMemcpy(dev_va, A.v, A.n * A.m * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_vb, B.v, B.n * B.m * sizeof(double), hipMemcpyHostToDevice);
    kernelMatrixMul << <nblocks, nthreads >> > (dev_va, dev_vb, A.n, A.m, B.m, dev_vdst);
    matrix_t productAB_GPU;
    matrix_alloc(A.n, B.m, &productAB_GPU);
    hipMemcpy(productAB_GPU.v, dev_vdst, A.n * B.m * sizeof(double), hipMemcpyDeviceToHost);

    /* Monitoriza */
    hipEventRecord(global_end);
    hipEventSynchronize(global_end);
    float gpu_ms;
    hipEventElapsedTime(&gpu_ms, global_start, global_end);
    printf("t_gpu_s:%.8f\n", gpu_ms/1000.0f);

    if (productAB_GPU.n * productAB_GPU.m < 9 * 9) {
        printf("Producto de ambas matrices (GPU):\n");
        matrix_print(productAB_GPU);
    }
    assert(memcmp(productAB.v, productAB_GPU.v, productAB.n * productAB.m * sizeof(double)) == 0);


    /* Destruye los recursos utilizados */
    matrix_destroy(&A);
    matrix_destroy(&B);
    matrix_destroy(&productAB);
    matrix_destroy(&productAB_GPU);
    hipFree(dev_va);
    hipFree(dev_vb);
    hipFree(dev_vdst);

    return 0;
}