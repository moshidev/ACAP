#include "hip/hip_runtime.h"



#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <stdint.h>
#include <locale.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#ifdef WIN32
#include <Windows.h>
int gettimeofday(struct timeval* tp, struct timezone* tzp)
{
    // Note: some broken versions only have 8 trailing zero's, the correct epoch has 9 trailing zero's
    // This magic number is the number of 100 nanosecond intervals since January 1, 1601 (UTC)
    // until 00:00:00 January 1, 1970 
    static const uint64_t EPOCH = ((uint64_t)116444736000000000ULL);

    SYSTEMTIME  system_time;
    FILETIME    file_time;
    uint64_t    time;

    GetSystemTime(&system_time);
    SystemTimeToFileTime(&system_time, &file_time);
    time = ((uint64_t)file_time.dwLowDateTime);
    time += ((uint64_t)file_time.dwHighDateTime) << 32;

    tp->tv_sec = (long)((time - EPOCH) / 10000000L);
    tp->tv_usec = (long)(system_time.wMilliseconds * 1000);
    return 0;
}
#else
#include <sys/time.h>
#endif

static double get_wall_time() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

#define NULL_PTR    0
#define STRTOL_INFER_BASE   0

typedef struct matrix {
    size_t n;
    size_t m;
    double* v;
} matrix_t;

#define acc(matrix, i, j) \
    ((matrix).v[i*(matrix).m+j])

int matrix_alloc(size_t n, size_t m, matrix_t* matrix) {
    double* vector = (double*)calloc(n*m, sizeof(double));
    if (vector) {
        matrix->n = n;
        matrix->m = m;
        matrix->v = vector;
        return 0;
    }
    return -1;
}

void matrix_destroy(matrix_t* matrix) {
    matrix->n = 0;
    matrix->m = 0;
    free(matrix->v);
    matrix->v = 0;
}

void matrix_print(matrix_t m) {
    for (size_t i = 0; i < m.n; i++) {
        for (size_t j = 0; j < m.m; j++) {
            printf("%.2f", acc(m, i, j));
            putchar(j != m.m-1 ? '\t' : '\n');
        }
    }
}

int matrix_mul(matrix_t A, matrix_t B, matrix_t* dst) {
    if (A.m != B.n) {
        return -1;
    }
    int errDst = matrix_alloc(A.n, B.m, dst);
    if (errDst != 0) {
        return -2;
    }

    const size_t suml = A.m;
    for (size_t i = 0; i < dst->n; i++) {
        for (size_t j = 0; j < dst->m; j++) {
            double sum = 0.0;
            for (size_t w = 0; w < suml; w++) {
                sum += acc(A, i, w) * acc(B, w, j);
            }
            acc(*dst, i, j) = sum;
        }
        /*if (i % (dst->n/100) == 0) {
            printf("%.2f%%\r", 100.0*i/dst->n);
            fflush(stdout);
        }*/
    }

    return 0;
}

void randomize(double* v, size_t v_len) {
    int seed = time(NULL_PTR);
    srand(seed);
    printf("Semilla utilizada: %d\n", seed);
    size_t p = 0;
    for (size_t i = 0; i < v_len; i++) {
        if (p >= 100000000) {
            p = 0;
            printf("Inicializados %zu elementos aleatorios (%f%%).\r", i, 100.0*((double)i)/v_len);
        }
        ++p;
        v[i] = rand()*rand();
    }
    printf("Inicializados %zu elementos aleatorios (%f%%).\n", v_len, 100.0);
}

void memExit39(void) {
    fprintf(stderr, "ª >w< son demasiados datos! no he podido reservar un trozo de memoria contigua tan grande!\n");
    fprintf(stderr, "Error fatal\n");
    exit(39);
}

__global__ void kernelMatrixMul(double* va, double* vb, size_t n, size_t m, size_t l, double* dst) {

}

int main(int argc, char** argv) {
    setlocale(LC_ALL, "");

    /* Inicializamos dos matrices */
    matrix_t A, B;
    int errA = matrix_alloc(3, 3, &A);
    int errB = matrix_alloc(3, 3, &B);
    if (errA != 0 || errB != 0) {
        memExit39();
    }
    //randomize(A.v, A.n*A.m);
    //randomize(B.v, B.n*B.m);
    for (size_t i = 0; i < A.n*A.m; i++) {
        A.v[i] = i%5+1;
        B.v[i] = i%5+1;
    }

    /* Multiplicamos ambas matrices en la CPU */
    double cpu_s = get_wall_time();
    matrix_t productAB;
    int errMul = matrix_mul(A, B, &productAB);
    if (errMul != 0) {
        memExit39();
    }
    if (productAB.n*productAB.m < 9*9) {
        printf("Matriz A:\n");
        matrix_print(A);
        printf("Matriz B:\n");
        matrix_print(B);
        printf("Producto de ambas matrices:\n");
        matrix_print(productAB);
    }
    cpu_s = get_wall_time() - cpu_s;
    printf("t_cpu_s:%.8f\n", cpu_s);

    /* Multiplicamos ambas matrices en la GPU */


    /* Destruye los recursos utilizados */
    matrix_destroy(&A);
    matrix_destroy(&B);
    matrix_destroy(&productAB);

    return 0;
}