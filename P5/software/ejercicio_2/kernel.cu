#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <stdint.h>
#include <locale.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#ifdef WIN32
#include <Windows.h>
int gettimeofday(struct timeval* tp, struct timezone* tzp)
{
    // Note: some broken versions only have 8 trailing zero's, the correct epoch has 9 trailing zero's
    // This magic number is the number of 100 nanosecond intervals since January 1, 1601 (UTC)
    // until 00:00:00 January 1, 1970 
    static const uint64_t EPOCH = ((uint64_t)116444736000000000ULL);

    SYSTEMTIME  system_time;
    FILETIME    file_time;
    uint64_t    time;

    GetSystemTime(&system_time);
    SystemTimeToFileTime(&system_time, &file_time);
    time = ((uint64_t)file_time.dwLowDateTime);
    time += ((uint64_t)file_time.dwHighDateTime) << 32;

    tp->tv_sec = (long)((time - EPOCH) / 10000000L);
    tp->tv_usec = (long)(system_time.wMilliseconds * 1000);
    return 0;
}
#else
#include <sys/time.h>
#endif

static double get_wall_time() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

#define NULL_PTR    0
#define STRTOL_INFER_BASE   0

static bool is_power_of_two(int n) {
    if (n <= 0) {
        return false;
    }

    return ceil(log2(n)) == floor(log2(n));
}

static void assert_good_args(int argc, char** argv) {
    const char* bad_input_help_text = "No puedo ejecutar el programa si no me indicas ni el número de hebras ni de qué tamaño quieres el array de números aleatorios. "
        "El tamaño del vector debe ser múltiplo del tamaño de bloque. "
        "El número de hebras debe ser múltiplo de dos.\n";
    const char* usage_help_text = "Uso: %s [número de hebras] [longitud]\n";
    if (argc != 3) {
        fprintf(stderr, "%s", bad_input_help_text);
        fprintf(stderr, usage_help_text, argv[0]);
        exit(1);
    }

    size_t nthreads = strtol(argv[1], NULL_PTR, STRTOL_INFER_BASE);
    size_t vector_len = strtol(argv[2], NULL_PTR, STRTOL_INFER_BASE);
    if (nthreads == 0 || vector_len == 0 || vector_len % nthreads != 0 || nthreads > 512 || !is_power_of_two(nthreads)) {
        fprintf(stderr, "%s", bad_input_help_text);
        fprintf(stderr, usage_help_text, argv[0]);
        fprintf(stderr, "Leído: %s %zu %zu", argv[0], nthreads, vector_len);
        exit(2);
    }
}

__global__ void maxv(double* v, size_t vlen, double* block_max) {
    extern __shared__ double sdata[];
    const unsigned tid = threadIdx.x;
    const unsigned gridSize = blockDim.x * gridDim.x;
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = -INFINITY;
    if (i < vlen) {
        sdata[tid] = v[i];
        i += gridSize;
    }
    while (i < vlen) {
        sdata[tid] = v[i] > sdata[tid] ? v[i] : sdata[tid];
        i += gridSize;
    }
    __syncthreads();

    for (unsigned s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = sdata[tid + s] > sdata[tid] ? sdata[tid + s] : sdata[tid];
        }
        __syncthreads();
    }

    if (tid == 0) {
        block_max[blockIdx.x] = sdata[0];
    }
}

size_t calc_max_idx_seq(const double* v, size_t vlen) {
    double max = -INFINITY;
    size_t max_i = (size_t) - 1;
    for (size_t i = 0; i < vlen; i++) {
        if (v[i] > max) {
            max = v[i];
            max_i = i;
        }
    }
    return max_i;
}

int main(int argc, char** argv) {
    setlocale(LC_ALL, "");

    /* Comprueba los parámetros introducidos, abortando si no son correctos */
    assert_good_args(argc, argv);

    /* Leemos los argumentos que se nos proporcionan */
    int nthreads = strtol(argv[1], NULL_PTR, STRTOL_INFER_BASE);
    size_t vector_len = strtol(argv[2], NULL_PTR, STRTOL_INFER_BASE);
    int nblocks = 3072;

    /* Inicializamos un vector de double valores aleatorios */
    int seed = time(NULL_PTR);
    srand(seed);
    printf("Semilla utilizada: %d\n", seed);
    double* vector = (double*)calloc(vector_len, sizeof(double));
    if (!vector) {
        fprintf(stderr, "ª >w< son demasiados datos! no he podido reservar un trozo de memoria contigua tan grande!\n");
        fprintf(stderr, "Error fatal\n");
        exit(39);
    }
    size_t p = 0;
    for (size_t i = 0; i < vector_len; i++) {
        if (p >= 100000000) {
            p = 0;
            printf("Inicializados %zu elementos (%f%%).\r", i, 100.0*((double)i)/vector_len);
        }
        ++p;
        vector[i] = rand()*rand();
    }
    printf("Inicializados %zu elementos (%f%%).\n", vector_len, 100.0);

    /* Monitoriza */
    hipEvent_t global_start, global_end;
    hipEventCreate(&global_start);
    hipEventCreate(&global_end);
    hipEventRecord(global_start);

    /* Lanza kernel */
    double* dev_vector;
    double* dev_block_max;
    hipMalloc(&dev_vector, vector_len*sizeof(double));
    hipMalloc(&dev_block_max, nblocks*sizeof(double));
    hipMemcpy(dev_vector, vector, vector_len * sizeof(double), hipMemcpyHostToDevice);

    maxv << <nblocks, nthreads, nthreads * sizeof(double) >> > (dev_vector, vector_len, dev_block_max);
    maxv << <1, nthreads, nthreads * sizeof(double) >> > (dev_block_max, nblocks, dev_block_max);

    double cuda_max;
    hipMemcpy(&cuda_max, dev_block_max, 1 * sizeof(double), hipMemcpyDeviceToHost);

    /* Monitoriza */
    hipEventRecord(global_end);
    hipEventSynchronize(global_end);

    hipDeviceSynchronize();
    double cpu_s = get_wall_time();
    double mmm = vector[calc_max_idx_seq(vector, vector_len)];
    cpu_s = get_wall_time() - cpu_s;
    printf("Esperado %f obtenido %f\n", mmm, cuda_max);

    /* Imprime los resultados de la ejecución */
    float elapsed_ms;
    hipEventElapsedTime(&elapsed_ms, global_start, global_end);
    printf("Tiempo total de ejecución del kernel<<<%d, %d>>> sobre %zu números en coma flotante de precisión doble [ms]: %.8f\n", nblocks, nthreads, vector_len, elapsed_ms);
    printf("Tiempo de CPU [ms] %f, Speedup %f\n", cpu_s * 1000.0, cpu_s * 1000.0 / elapsed_ms);
    assert(mmm == cuda_max);

    /* Destruye los recursos utilizados */
    hipEventDestroy(global_start);
    hipEventDestroy(global_end);
    hipFree(dev_block_max);
    hipFree(dev_vector);
    free(vector);

    return 0;
}